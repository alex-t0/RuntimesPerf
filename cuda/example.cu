#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <cmath>

// Error checking macro for CUDA API calls
#define CUDA_CHECK(call) \
    do { \
        hipError_t err = call; \
        if (err != hipSuccess) { \
            std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
            exit(EXIT_FAILURE); \
        } \
    } while (0)

// CUDA kernel for square root calculation
__global__ void sqrtKernel(int* input, double* output, int size, double precision) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= size) return;

    int num = input[idx];
    if (num == 0) {
        output[idx] = 0.0;
        return;
    }

    // CalculateEstimation logic
    int copyForCountDigits = num;
    int countDigits = 0;
    int firstTwoDigits[2] = { 0, 0 };

    while (copyForCountDigits > 0) {
        firstTwoDigits[1] = firstTwoDigits[0];
        firstTwoDigits[0] = copyForCountDigits;
        copyForCountDigits /= 10;
        countDigits++;
    }

    double estimation;
    if (firstTwoDigits[0] == 0 && firstTwoDigits[1] == 0) {
        estimation = 0.0;
    }
    else {
        int rootBase = firstTwoDigits[0];
        int exponent = countDigits - 1;

        if (countDigits % 2 == 0) {
            rootBase = firstTwoDigits[1];
            exponent = countDigits - 2;
        }

        estimation = rootBase < 10 ? 2.0 * pow(10.0, exponent / 2) : 6.0 * pow(10.0, exponent / 2);
    }

    // Sqrt logic (Newton-Raphson iteration)
    double currentPrecision = 1.0;
    while (currentPrecision > precision) {
        estimation = (num / estimation + estimation) / 2.0;
        currentPrecision = fabs(estimation * estimation - num);
    }

    output[idx] = estimation;
}

int main() {
    // Parameters
    const int N = 5000000; // Same as original
    const double precision = 0.00000001; // Same as original
    const int blockSize = 512; // CUDA threads per block
    const int gridSize = (N + blockSize - 1) / blockSize; // CUDA blocks

    // Host arrays
    int* h_input = new int[N];
    double* h_output = new double[N];

    // Initialize input array (0 to N-1)
    for (int i = 0; i < N; ++i) {
        h_input[i] = i;
    }

    // Device arrays
    int* d_input;
    double* d_output;
    CUDA_CHECK(hipMalloc(&d_input, N * sizeof(int)));
    CUDA_CHECK(hipMalloc(&d_output, N * sizeof(double)));

    // Copy input data to device
    CUDA_CHECK(hipMemcpy(d_input, h_input, N * sizeof(int), hipMemcpyHostToDevice));

    // Warm-up run
    sqrtKernel << <gridSize, blockSize >> > (d_input, d_output, N, precision);
    CUDA_CHECK(hipDeviceSynchronize());

    // Benchmarking
    auto start = std::chrono::high_resolution_clock::now();
    sqrtKernel << <gridSize, blockSize >> > (d_input, d_output, N, precision);
    CUDA_CHECK(hipDeviceSynchronize());
    auto finish = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> elapsed = finish - start;

    // Copy results back to host
    CUDA_CHECK(hipMemcpy(h_output, d_output, N * sizeof(double), hipMemcpyDeviceToHost));

    // Output results (same as original)
    std::cout.precision(17);
    std::cout << "4250250 item: " << h_output[4250250] << std::endl;
    std::cout << "Elapsed " << elapsed.count() << std::endl;

    // Cleanup
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    delete[] h_input;
    delete[] h_output;

    return 0;
}
